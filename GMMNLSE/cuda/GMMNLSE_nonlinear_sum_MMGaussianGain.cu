
#include <hip/hip_runtime.h>
#define MAX_NUM_MODES 32 // the maximum number of modes for this cuda = sqrt(MaxThreadsPerBlock)
                         //                                           = sqrt(1024) for our Titan XP GPU

__global__ void GMMNLSE_nonlinear_sum_MMGaussianGain(double2* Kerr, double2* Ra, double2* transfer_matrix,
                                                     const double2* A_t,
                                                     const double2* Bmn,
                                                     const double* SK, const double* SRa,
                                                     const unsigned char* nonzero_midx1234s,
                                                     const unsigned int* beginning_nonzero, const unsigned int* ending_nonzero,
                                                     const bool include_Raman,
                                                     const unsigned int N, const unsigned int M,
                                                     const unsigned int NUM_MODES,
                                                     const unsigned int NUM_OPERATIONS) {
    const unsigned int midx1 = threadIdx.x / NUM_MODES;
    const unsigned int midx2 = threadIdx.x - midx1*NUM_MODES;

    const unsigned int NMIdx = blockIdx.x / NUM_OPERATIONS;
    const unsigned int OPERATIONIdx = blockIdx.x - NMIdx*NUM_OPERATIONS;

    const unsigned int Midx = NMIdx / N;
    const unsigned int Nidx = NMIdx - Midx*N;

    const unsigned int NM = N*M;
    const unsigned int NMMODES = NM*NUM_MODES;

    // Preload A_t to improve the performance (avoiding accessing the global memory too many times)
    __shared__ double2 this_A[MAX_NUM_MODES];
    if (midx1 == 0) this_A[midx2] = A_t[Nidx+Midx*N+midx2*NM];
    __syncthreads();

    const unsigned int this_beginning_nonzero = beginning_nonzero[midx2+midx1*NUM_MODES];
    const unsigned int this_ending_nonzero = ending_nonzero[midx2+midx1*NUM_MODES];

    unsigned int midx3, midx4;
    double a, b, c, d, e, f, pcdef;
    switch (OPERATIONIdx) {
        case 0: // compute the Kerr term
            if (this_beginning_nonzero > 0) {
                a = this_A[midx2].x;
                b = this_A[midx2].y;

                double2 this_Kerr;
                this_Kerr.x = 0; this_Kerr.y = 0; // initialized
                for (int i = this_beginning_nonzero-1; i < this_ending_nonzero-1; i++) {
                    midx3 = nonzero_midx1234s[2+i*4]-1;
                    midx4 = nonzero_midx1234s[3+i*4]-1;
                    
                    c = this_A[midx3].x;
                    d = this_A[midx3].y;
                    e = this_A[midx4].x;
                    f = this_A[midx4].y;
		            
                    pcdef = SK[i]*(c*e+d*f);
                    if (midx3 == midx4) { // d*e-c*f= 0
                        this_Kerr.x += a*pcdef;
                        this_Kerr.y += b*pcdef;
                    } else { // (d*e-c*f) + (c <--> e, d <--> f) = 0
                        this_Kerr.x += a*pcdef*2;
                        this_Kerr.y += b*pcdef*2;
                    }
                }
                Kerr[Nidx+Midx*N+midx1*NM+midx2*NMMODES] = this_Kerr;
            }
            break;

        case 1: // compute the Raman term
            if (this_beginning_nonzero > 0 && include_Raman) {
                double2 this_Ra;
                this_Ra.x = 0; this_Ra.y = 0; // initialized
                for (int i = this_beginning_nonzero-1; i < this_ending_nonzero-1; i++) {
                    midx3 = nonzero_midx1234s[2+i*4]-1;
                    midx4 = nonzero_midx1234s[3+i*4]-1;
            
                    c = this_A[midx3].x;
                    d = this_A[midx3].y;
                    e = this_A[midx4].x;
                    f = this_A[midx4].y;
                    
                    if (midx3 == midx4) { // d*e-c*f= 0
                        this_Ra.x += SRa[i]*(c*e+d*f);
                    } else { // (d*e-c*f) + (c <--> e, d <--> f) = 0
                        this_Ra.x += SRa[i]*(c*e+d*f)*2;
                    }
                }
                Ra[Nidx+Midx*N+midx1*NM+midx2*NMMODES] = this_Ra;
            }
            break;

        case 2: // compute the transfer matrix
            if (this_beginning_nonzero > 0 && Nidx == 0) { // "Nidx==0" is to limit this to run only once through different Nidx
                double2 this_transfer_matrix;
                this_transfer_matrix.x = 0; this_transfer_matrix.y = 0; // initialized
                for (int i = this_beginning_nonzero-1; i < this_ending_nonzero-1; i++) {
                    midx3 = nonzero_midx1234s[2+i*4]-1;
                    midx4 = nonzero_midx1234s[3+i*4]-1;

                    if (midx3 == midx4) {
                        this_transfer_matrix.x += SRa[i]*Bmn[Midx+midx3*M+midx4*M*NUM_MODES].x;
                        this_transfer_matrix.y += SRa[i]*Bmn[Midx+midx3*M+midx4*M*NUM_MODES].y;
                    } else {
                        this_transfer_matrix.x += SRa[i]*(Bmn[Midx+midx3*M+midx4*M*NUM_MODES].x+Bmn[Midx+midx4*M+midx3*M*NUM_MODES].x);
                        this_transfer_matrix.y += SRa[i]*(Bmn[Midx+midx3*M+midx4*M*NUM_MODES].y+Bmn[Midx+midx4*M+midx3*M*NUM_MODES].y);
                    }
                }
                transfer_matrix[Midx+midx1*M+midx2*M*NUM_MODES] = this_transfer_matrix;
            }
            break;
    }
}
