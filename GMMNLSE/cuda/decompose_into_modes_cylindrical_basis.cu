
#include <hip/hip_runtime.h>
__global__ void decompose_into_modes_cylindrical_basis(double2* field_wm,
                                                       const double* normalized_mode_space_profiles_xym, const double* norm_spatial_modes, const double2* full_field_xyw,
                                                       const double* r, const double dr, const double dtheta,
                                                       const unsigned int sM, const unsigned int sX, const unsigned int sY, const unsigned int sW) {
    unsigned int thread_idx = threadIdx.x + blockIdx.x*blockDim.x;

    if (threadIdx.x >= (sX*sY)) return;
    if (blockIdx.x >= sW) return;
    if (thread_idx >= (sX*sY*sW)) return;

    const unsigned int ri = threadIdx.x%sX;

    const unsigned int max_num_r = 100;
    const unsigned int max_sXsYsM = 1024*20;
    __shared__ double2 sum_field[max_sXsYsM]; //[sX*sY*sM]
    __shared__ double this_r[max_num_r];
    if (threadIdx.x < sX)
        this_r[threadIdx.x] = r[threadIdx.x];
    __syncthreads();

    // blockDim.x = sX*sY
    for (unsigned int mi = 0; mi<sM; mi++) {
        sum_field[threadIdx.x+blockDim.x*mi].x = normalized_mode_space_profiles_xym[threadIdx.x+blockDim.x*mi]*full_field_xyw[thread_idx].x*this_r[ri];
        sum_field[threadIdx.x+blockDim.x*mi].y = normalized_mode_space_profiles_xym[threadIdx.x+blockDim.x*mi]*full_field_xyw[thread_idx].y*this_r[ri];
    }
    __syncthreads();

    // sM <= blockDim.x = sX*sY
    // Each thread in each block deals with the summation of each mode with its corresponding frequency now,
    // that is, thread_idx = mi.
    if (threadIdx.x < sM) {
        for (unsigned int i = 0; i<blockDim.x; i++) {
            field_wm[blockIdx.x+sW*threadIdx.x].x = field_wm[blockIdx.x+sW*threadIdx.x].x + sum_field[i+blockDim.x*threadIdx.x].x;
            field_wm[blockIdx.x+sW*threadIdx.x].y = field_wm[blockIdx.x+sW*threadIdx.x].y + sum_field[i+blockDim.x*threadIdx.x].y;
        }
        field_wm[blockIdx.x+sW*threadIdx.x].x = field_wm[blockIdx.x+sW*threadIdx.x].x*dr*dtheta*norm_spatial_modes[threadIdx.x];
        field_wm[blockIdx.x+sW*threadIdx.x].y = field_wm[blockIdx.x+sW*threadIdx.x].y*dr*dtheta*norm_spatial_modes[threadIdx.x];
    }
}