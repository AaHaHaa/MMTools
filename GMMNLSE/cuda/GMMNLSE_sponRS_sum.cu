
#include <hip/hip_runtime.h>
#define MAX_NUM_MODES 32 // the maximum number of modes for this cuda = sqrt(MaxThreadsPerBlock)
                         //                                           = sqrt(1024) for our Titan XP GPU

__global__ void GMMNLSE_sponRS_sum(double2* Ra,
                                   const double2* A_t, const double2* A_t_sponRS,
                                   const double* SRa,
                                   const unsigned char* nonzero_midx1234s,
                                   const unsigned int* beginning_nonzero, const unsigned int* ending_nonzero,
                                   const unsigned int N, const unsigned int M,
                                   const unsigned int NUM_MODES) {
    const unsigned int midx1 = threadIdx.x / NUM_MODES;
    const unsigned int midx2 = threadIdx.x - midx1*NUM_MODES;

    const unsigned int Midx = blockIdx.x / N;
    const unsigned int Nidx = blockIdx.x - Midx*N;

    const unsigned int NM = N*M;
    const unsigned int NMMODES = NM*NUM_MODES;

    // Preload A_t to improve the performance (avoiding accessing the global memory too many times)
    __shared__ double2 this_A[MAX_NUM_MODES], this_A_sponRS[MAX_NUM_MODES];
    if (midx1 == 0) {
        this_A[midx2] = A_t[Nidx+Midx*N+midx2*NM];
        this_A_sponRS[midx2] = A_t_sponRS[Nidx+Midx*N+midx2*NM];
    }
    __syncthreads();

    const unsigned int this_beginning_nonzero = beginning_nonzero[midx2+midx1*NUM_MODES];
    const unsigned int this_ending_nonzero = ending_nonzero[midx2+midx1*NUM_MODES];

    unsigned int midx3, midx4;
    double c, d, e, f; // this_A
    double p, q, r, s; // this_A_sponRS
    // compute the spontaneous Raman term
    if (this_beginning_nonzero > 0) {
        double2 this_Ra;
        this_Ra.x = 0; this_Ra.y = 0; // initialized
        for (int i = this_beginning_nonzero-1; i < this_ending_nonzero-1; i++) {
            midx3 = nonzero_midx1234s[2+i*4]-1;
            midx4 = nonzero_midx1234s[3+i*4]-1;

            c = this_A[midx3].x;
            d = this_A[midx3].y;
            e = this_A[midx4].x;
            f = this_A[midx4].y;

            p = this_A_sponRS[midx3].x;
            q = this_A_sponRS[midx3].y;
            r = this_A_sponRS[midx4].x;
            s = this_A_sponRS[midx4].y;

            if (midx3 == midx4) {
                this_Ra.x += SRa[i]*( (p*r+q*s)   + (c*r+d*s)*2 );
            } else {
                this_Ra.x += SRa[i]*( (p*r+q*s)*2 + (c*r+d*s)*2+(e*p+f*q)*2 );
            }
        }
        Ra[Nidx+Midx*N+midx1*NM+midx2*NMMODES] = this_Ra;
    }
}